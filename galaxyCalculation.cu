#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <algorithm>  // For std::sort
#include <vector>     // For std::vector and std::pair
#include <limits.h>
#include <fstream>  // Include for file output
#include <iomanip> // Required for std::fixed and std::setprecision

#define BIN_RANGE 90 // Doesn't work with 180?
#define BINS_PER_DEGREE 4
#define THREADS_PER_BLOCK 256
#define RAD_TO_DEG 57.29577951

__global__ void calculateAngle(float* ra_A, float* decl_A, float* ra_B, float* decl_B, int* histogram, int N){
	float theta_deg;
	
	int tid = (blockDim.x * blockIdx.x + threadIdx.x);  // thread ID
	int k = tid % N;  // index of B
	int i = (tid - k) / N;  // index of A 

	if (i > N || k > N) {  // check that we're in bounds
		printf("Not in bounds\n");
		return;
	}

	// helper variables
	float alpha_A = ra_A[i];
	float delta_A = decl_A[i];
	float alpha_B = ra_B[k];
	float delta_B = decl_B[k];

	// TODO: check that we have logical values

	float dotProduct = cos(delta_A)*cos(delta_B)*cos(alpha_A-alpha_B)+sin(delta_A)*sin(delta_B);
	// dotProduct = fminf(1.0f, fmaxf(dotProduct, -1.0f)); // Clamp to [-1, 1]

	float theta_rad = acosf(dotProduct);
	theta_deg = theta_rad * RAD_TO_DEG;
	if (theta_deg < 0.0f){
		// Acosf should return positive values only lol
		printf("theta_deg is somehow negative?!?\n");
	}
	if (theta_deg > 180.0f){
		printf("theta_deg is larger than 180 degrees?!?\n");
	}
	// Remove floating point errors by clamping to [0, 180]
	// theta_deg = fminf(180.0f, fmaxf(theta_deg, 0.0f)); 

	// histogram time!
	int histogramBinIndex = theta_deg * BINS_PER_DEGREE; // Get the index by multiplying the angle by bins per degree

	if (histogramBinIndex >= 0 && histogramBinIndex < (BIN_RANGE * BINS_PER_DEGREE)){ // ensure boundary
		// TODO: maybe change to atomicInc, but then we need to change to size_t type arrays or something
		atomicAdd(&histogram[histogramBinIndex], 1);  // incementing histograms now, probably slows down the execution time a lot, 
		// but we don't have to launch another GPU program
	}
}

void verbose_omega(float* omega){
	printf("\nThe first five bins: \n");
	for (int i=0; i<5; i++){
		printf("Bin [%d] value: 	 %.5f\n", i, omega[i]);
	}

	// Prepare vector of (bin index, value) pairs for sorting
    std::vector<std::pair<int, float>> bin_counts;
    for (int i = 0; i < BIN_RANGE*BINS_PER_DEGREE; i++) {
        bin_counts.push_back(std::make_pair(i, omega[i]));
    }

	// Sort by count in descending order for top 3, ascending for bottom 3
    std::sort(bin_counts.begin(), bin_counts.end(), [](const auto &a, const auto &b) { return a.second > b.second; });

    // Top 3 bins with the most entries
    printf("\nTop 3 most populated bins:\n");
    for (int j = 0; j < 3 && j < bin_counts.size(); j++) {
        int bin_idx = bin_counts[j].first;
        float count = bin_counts[j].second;
        float angle = (float)bin_idx / BINS_PER_DEGREE;  // Convert bin index to angle in degrees
        printf("Bin %d (Angle ≈ %.2f°): Value = %.5f\n", bin_idx, angle, count);
    }
}

void verbose_histogram(int* histogram) {
    long total_count = 0;
    int max_height = 0;
    int mode_bin = 0;
    int non_zero_start = -1, non_zero_end = -1;
    int NUM_BINS = BIN_RANGE * BINS_PER_DEGREE;

    // Calculate total count, find max height and mode, and locate non-zero range
    for (int i = 0; i < NUM_BINS; i++) {
        total_count += histogram[i];
        
        if (histogram[i] > max_height) {
            max_height = histogram[i];
            mode_bin = i;
        }
        
        if (histogram[i] > 0) {
            if (non_zero_start == -1) non_zero_start = i;
            non_zero_end = i;
        }
    }
    
    // Mean and standard deviation
    double mean_height = (double)total_count / NUM_BINS;
    double variance = 0.0;
    for (int i = 0; i < NUM_BINS; i++) {
        variance += pow(histogram[i] - mean_height, 2);
    }
    variance /= NUM_BINS;
    double stddev = sqrt(variance);

    // Prepare vector of (bin index, count) pairs for sorting
    std::vector<std::pair<int, int>> bin_counts;
    for (int i = 0; i < NUM_BINS; i++) {
        bin_counts.push_back(std::make_pair(i, histogram[i]));
    }

	printf("\nThe first five bins: \n");
	for (int i=0; i<5; i++){
		printf("Bin [%d] count: 	 %d\n", i, histogram[i]);
	}

    // Sort by count in descending order for top 3, ascending for bottom 3
    std::sort(bin_counts.begin(), bin_counts.end(), [](const auto &a, const auto &b) { return a.second > b.second; });

    // Top 3 bins with the most entries
    printf("\nTop 3 most populated bins:\n");
    for (int j = 0; j < 3 && j < bin_counts.size(); j++) {
        int bin_idx = bin_counts[j].first;
        int count = bin_counts[j].second;
        float angle = (float)bin_idx / BINS_PER_DEGREE;  // Convert bin index to angle in degrees
        printf("Bin %d (Angle ≈ %.2f°): Count = %d\n", bin_idx, angle, count);
    }

    // Sort by ascending order to find bottom 3 non-zero populated bins
    std::sort(bin_counts.begin(), bin_counts.end(), [](const auto &a, const auto &b) { return a.second < b.second; });

    // Bottom 3 bins with the least entries (excluding zero counts)
    printf("\nBottom 3 least populated bins (non-zero):\n");
    int count_bottom = 0;
    for (const auto& bin : bin_counts) {
        if (bin.second > 0) {  // Exclude zero entries
            int bin_idx = bin.first;
            int count = bin.second;
            float angle = (float)bin_idx / BINS_PER_DEGREE;  // Convert bin index to angle in degrees
            printf("Bin %d (Angle ≈ %.2f°): Count = %d\n", bin_idx, angle, count);
            count_bottom++;
            if (count_bottom >= 3) break;  // Only get bottom 3
        }
    }

    // Output summary
    printf("\nHistogram Summary:\n");
    printf("Total count: %ld\n", total_count);
    printf("Mean bin height: %.2f\n", mean_height);
    printf("Mode bin: %d with height %d\n", mode_bin, max_height);
    printf("Standard deviation of bin heights: %.2f\n", stddev);
    printf("Non-zero bin range: %d to %d\n", non_zero_start, non_zero_end);
}

void save_histogram_to_file(const int* histogram, int num_bins, const char* filename) {
    std::ofstream outfile(filename);
    if (!outfile.is_open()) {
        printf("Error opening file %s for writing.\n", filename);
        return;
    }
    for (int i = 0; i < num_bins; i++) {
        outfile << i << " " << histogram[i] << "\n";
    }
    outfile.close();
    printf("Histogram saved to %s\n", filename);
}


// data for the real galaxies will be read into these arrays
float *h_phiReal, *h_thetaReal;
// number of real galaxies
int nrReal;

// data for the simulated random galaxies will be read into these arrays
float *h_phiFake, *h_thetaFake;
// number of simulated random galaxies
int nrFake;

int *histogramDR, *histogramDD, *histogramRR;
int *d_histogram;

int main(int argc, char *argv[])
{
	printf("==================== READING INPUT DATA =====================\n");
	int readdata(char *argv1, char *argv2);
	printf("==================== READING DEVICE DATA ====================\n");
	int getDevice(int deviceNr);

	if (argc != 4)
	{
		printf("Usage: a.out real_data random_data output_data\n");
		return (-1);
	}

	if (getDevice(0) != 0){
		printf("Failed finding a device!");
		return (-1);
	}
		

	if (readdata(argv[1], argv[2]) != 0){
		printf("Failed reading data!");
		return (-1);
	}
		
	// make sure input array sizes are the same
	
	int N = nrReal;
	if (N != nrFake){
		printf("Input data lengths are not the same! Exit...");
		return (-1);
	}

	printf("N = %d\n", N);

	clock_t start, end;
	double time_used;
	start = clock();

	long int totalPairs = (long int)N*N;

	// allocate memory on the GPU and histogram arrays memory on CPU
	size_t arraybytes = N * sizeof(float);
	printf("Arraybytes: %d\n", arraybytes);

	size_t histogrambytes = BIN_RANGE * BINS_PER_DEGREE * sizeof(int);
	int* h_histogramDR = (int*)malloc(histogrambytes);
	int* h_histogramDD = (int*)malloc(histogrambytes);
	int* h_histogramRR = (int*)malloc(histogrambytes);
	
	// allocate to GPU: the real and fake right ascension and declination
	float* d_phiReal; hipMalloc(&d_phiReal, arraybytes);
	float* d_thetaReal; hipMalloc(&d_thetaReal, arraybytes);
	float* d_phiFake; hipMalloc(&d_phiFake, arraybytes);
	float* d_thetaFake; hipMalloc(&d_thetaFake, arraybytes);

	// copy data to the GPU
	hipMemcpy(d_phiReal, h_phiReal, arraybytes, hipMemcpyHostToDevice);
	hipMemcpy(d_thetaReal, h_thetaReal, arraybytes, hipMemcpyHostToDevice);
	hipMemcpy(d_phiFake, h_phiFake, arraybytes, hipMemcpyHostToDevice);
	hipMemcpy(d_thetaFake, h_thetaFake, arraybytes, hipMemcpyHostToDevice);

	// Size of thread blocks
	int blocksInGrid = (totalPairs + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	printf("Blocks in grid: %d\n", blocksInGrid);

	// run the kernels on the GPU
	// THIS CALCULATES DR
	// 1. Calculate Real vs. Fake (DR histogram) - already implemented
    printf("================= CALCULATING DR ANGLES =====================\n");
	int *d_histogramDR;
	hipMalloc(&d_histogramDR, histogrambytes);
	hipMemset(d_histogramDR, 0, histogrambytes);
    calculateAngle<<<blocksInGrid, THREADS_PER_BLOCK>>>(d_phiReal, d_thetaReal, d_phiFake, d_thetaFake, d_histogramDR, N);
	memset(h_histogramDR, 0, histogrambytes);
    hipMemcpy(h_histogramDR, d_histogramDR, histogrambytes, hipMemcpyDeviceToHost);
	hipFree(d_histogramDR); 

    // 2. Calculate Real vs. Real (DD histogram)
    printf("================= CALCULATING DD ANGLES =====================\n");
	int *d_histogramDD;
	hipMalloc(&d_histogramDD, histogrambytes);
	hipMemset(d_histogramDD, 0, histogrambytes);
    calculateAngle<<<blocksInGrid, THREADS_PER_BLOCK>>>(d_phiReal, d_thetaReal, d_phiReal, d_thetaReal, d_histogramDD, N);
	memset(h_histogramDD, 0, histogrambytes);
    hipMemcpy(h_histogramDD, d_histogramDD, histogrambytes, hipMemcpyDeviceToHost);
	hipFree(d_histogramDD); 

    // 3. Calculate Fake vs. Fake (RR histogram)
    printf("================= CALCULATING RR ANGLES =====================\n");
	int *d_histogramRR;
	hipMalloc(&d_histogramRR, histogrambytes);
	hipMemset(d_histogramRR, 0, histogrambytes);
    calculateAngle<<<blocksInGrid, THREADS_PER_BLOCK>>>(d_phiFake, d_thetaFake, d_phiFake, d_thetaFake, d_histogramRR, N);
	memset(h_histogramRR, 0, histogrambytes);
    hipMemcpy(h_histogramRR, d_histogramRR, histogrambytes, hipMemcpyDeviceToHost);
	hipFree(d_histogramRR);
	
	printf("DONE!\n");
	
	// Free memory
	hipFree(d_thetaReal); hipFree(d_phiReal); 
	hipFree(d_thetaFake); hipFree(d_phiFake);
	

	end = clock();
	time_used = ((double) (end - start)) / CLOCKS_PER_SEC * 1000.0;
	printf("Execution time: %.2f ms\n", time_used);
    
	printf("===================== CALCULATING OMEGA =====================\n");
	// calculate omega values on the CPU
	// Memory management
	size_t omegabytes = BIN_RANGE*BINS_PER_DEGREE*sizeof(float);
	float* h_omega = (float*)malloc(omegabytes);
	for (int i=0; i<BIN_RANGE*BINS_PER_DEGREE; i++){
		float num = (float)(h_histogramDD[i] - (2*h_histogramDR[i]) + h_histogramRR[i]);
		float den = (float)(h_histogramRR[i]);
		if (den == 0.0f){
			printf("Division by zero!\n");
		} else {
			h_omega[i] = (float)(num/den);
		}
	}

	// write omega values to omega.out
	printf("=================== SAVING OMEGA VALUES =====================\n");
	std::ofstream outfile("omega.txt");
    if (!outfile.is_open()) {
        printf("Error opening file %s for writing.\n", "omega.txt");
        return 0;
    }

    for (int i = 0; i < BIN_RANGE*BINS_PER_DEGREE; i++) {
        outfile << i << " " << h_omega[i] << "\n";
    }
    outfile.close();
    printf("Omega values saved to %s\n", "omega.txt");

	printf("====================== OMEGA SUMMARY ========================\n");

	verbose_omega(h_omega);

	printf("\n");
	printf("================== SUMMARIZING HISTOGRAMS ===================\n");
	printf("Summary for Real vs. Fake (DR):\n");
    verbose_histogram(h_histogramDR);
	printf("\n");
    printf("Summary for Real vs. Real (DD):\n");
    verbose_histogram(h_histogramDD);
	printf("\n");
    printf("Summary for Fake vs. Fake (RR):\n");
    verbose_histogram(h_histogramRR);
	printf("\n");

	printf("===================== SAVING HISTOGRAMS =====================\n");
	// save the histogram to a file for analyzing later
	save_histogram_to_file(h_histogramDR, BIN_RANGE * BINS_PER_DEGREE, "histogramDR.txt");
	save_histogram_to_file(h_histogramDD, BIN_RANGE * BINS_PER_DEGREE, "histogramDD.txt");
	save_histogram_to_file(h_histogramRR, BIN_RANGE * BINS_PER_DEGREE, "histogramRR.txt");

	printf("======================= DONE, GOODBYE! ======================\n");

	// Free host memory
	free(h_histogramDD); free(h_histogramDR); free(h_histogramRR); free(h_omega);

	return (0);
}

int readdata(char *argv1, char *argv2)
{
	int i, linecount;
	char inbuf[180];
	double ra, dec; // phi, theta, dpi;
	FILE *infil;

	printf("   Assuming input data is given in arc minutes!\n");

	float dpi = acos(-1.0);

	infil = fopen(argv1, "r");
	if (infil == NULL)
	{
		printf("Cannot open input file %s\n", argv1);
		return (-1);
	}

	// read the number of galaxies in the input file
	int announcednumber;
	if (fscanf(infil, "%d\n", &announcednumber) != 1)
	{
		printf(" cannot read file %s\n", argv1);
		return (-1);
	}
	linecount = 0;
	while (fgets(inbuf, 180, infil) != NULL)
		++linecount;
	rewind(infil);

	if (linecount == announcednumber)
		printf("   %s contains %d galaxies\n", argv1, linecount);
	else
	{
		printf("   %s does not contain %d galaxies but %d\n", argv1, announcednumber, linecount);
		return (-1);
	}

	nrReal = linecount;
	h_phiReal = (float *)calloc(nrReal, sizeof(float));
	h_thetaReal = (float *)calloc(nrReal, sizeof(float));

	// skip the number of galaxies in the input file
	if (fgets(inbuf, 180, infil) == NULL)
		return (-1);
	i = 0;
	while (fgets(inbuf, 80, infil) != NULL)
	{
		if (sscanf(inbuf, "%lf %lf", &ra, &dec) != 2)
		{
			printf("   Cannot read line %d in %s\n", i + 1, argv1);
			fclose(infil);
			return (-1);
		}
		// spherical coordinates phi and theta in radians:
		// phi   = ra/60.0 * dpi/180.0;
		// theta = (90.0-dec/60.0)*dpi/180.0;
		// store values as phi and theta in radians instead of right ascension and declination in arc minutes

		h_phiReal[i] = (float) (ra / 60.0f) * (dpi / 180.0f);
		h_thetaReal[i] = (float) (90.0f - dec / 60.0f) * (dpi / 180.0f);
		++i;
	}

	fclose(infil);

	if (i != nrReal)
	{
		printf("   Cannot read %s correctly\n", argv1);
		return (-1);
	}

	infil = fopen(argv2, "r");
	if (infil == NULL)
	{
		printf("Cannot open input file %s\n", argv2);
		return (-1);
	}

	if (fscanf(infil, "%d\n", &announcednumber) != 1)
	{
		printf(" cannot read file %s\n", argv2);
		return (-1);
	}
	linecount = 0;
	while (fgets(inbuf, 80, infil) != NULL)
		++linecount;
	rewind(infil);

	if (linecount == announcednumber)
		printf("   %s contains %d galaxies\n", argv2, linecount);
	else
	{
		printf("   %s does not contain %d galaxies but %d\n", argv2, announcednumber, linecount);
		return (-1);
	}

	nrFake = linecount;
	h_phiFake = (float *)calloc(nrFake, sizeof(float));
	h_thetaFake = (float *)calloc(nrFake, sizeof(float));

	// skip the number of galaxies in the input file
	if (fgets(inbuf, 180, infil) == NULL)
		return (-1);
	i = 0;
	while (fgets(inbuf, 80, infil) != NULL)
	{
		if (sscanf(inbuf, "%lf %lf", &ra, &dec) != 2)
		{
			printf("   Cannot read line %d in %s\n", i + 1, argv2);
			fclose(infil);
			return (-1);
		}
		// spherical coordinates phi and theta in radians:
		// phi   = ra/60.0 * dpi/180.0;
		// theta = (90.0-dec/60.0)*dpi/180.0;
		// store values as phi and theta in radians instead of right ascension and declination in arc minutes

		h_phiFake[i] = (float) (ra / 60.0f) * (dpi / 180.0f);
		h_thetaFake[i] = (float) (90.0f - dec / 60.0f) * (dpi / 180.0f);
		++i;
	}

	fclose(infil);

	if (i != nrFake)
	{
		printf("   Cannot read %s correctly\n", argv2);
		return (-1);
	}

	return (0);
}

int getDevice(int deviceNo)
{

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("   Found %d CUDA devices\n", deviceCount);
	if (deviceCount < 0 || deviceCount > 128)
		return (-1);
	int device;
	for (device = 0; device < deviceCount; ++device)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("      Device %s                  device %d\n", deviceProp.name, device);
		printf("         compute capability            =        %d.%d\n", deviceProp.major, deviceProp.minor);
		printf("         totalGlobalMemory             =       %.2lf GB\n", deviceProp.totalGlobalMem / 1000000000.0);
		printf("         l2CacheSize                   =   %8d B\n", deviceProp.l2CacheSize);
		printf("         regsPerBlock                  =   %8d\n", deviceProp.regsPerBlock);
		printf("         multiProcessorCount           =   %8d\n", deviceProp.multiProcessorCount);
		printf("         maxThreadsPerMultiprocessor   =   %8d\n", deviceProp.maxThreadsPerMultiProcessor);
		printf("         sharedMemPerBlock             =   %8d B\n", (int)deviceProp.sharedMemPerBlock);
		printf("         warpSize                      =   %8d\n", deviceProp.warpSize);
		printf("         clockRate                     =   %8.2lf MHz\n", deviceProp.clockRate / 1000.0);
		printf("         maxThreadsPerBlock            =   %8d\n", deviceProp.maxThreadsPerBlock);
		printf("         asyncEngineCount              =   %8d\n", deviceProp.asyncEngineCount);
		printf("         f to lf performance ratio     =   %8d\n", deviceProp.singleToDoublePrecisionPerfRatio);
		printf("         maxGridSize                   =   %d x %d x %d\n",
			   deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
		printf("         maxThreadsDim in thread block =   %d x %d x %d\n",
			   deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
		printf("         concurrentKernels             =   ");
		if (deviceProp.concurrentKernels == 1)
			printf("     yes\n");
		else
			printf("    no\n");
		printf("         deviceOverlap                 =   %8d\n", deviceProp.deviceOverlap);
		if (deviceProp.deviceOverlap == 1)
			printf("            Concurrently copy memory/execute kernel\n");
	}

	hipSetDevice(deviceNo);
	hipGetDevice(&device);
	if (device != deviceNo)
		printf("   Unable to set device %d, using device %d instead", deviceNo, device);
	else
		printf("   Using CUDA device %d\n\n", device);

	return (0);
}